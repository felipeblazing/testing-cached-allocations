#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : testers.cu
 Author      : BlazingDB
 Version     :
 Copyright   : Your copyright notice
 Description : Compute sum of reciprocals using STL on CPU and Thrust on GPU
 ============================================================================
 */

#include <algorithm>
#include <iostream>
#include <numeric>
#include <vector>
#include <ctime>

#include <thrust/reduce.h>
#include <thrust/device_vector.h>

#include "BlazingCachedAllocator.cuh"

template <typename T> __host__ __device__  T reciprocal(const T &x)
{
	return 2 * x;
}

template <typename T> class ReciprocalFunctor {
	public:
	__host__ __device__ T operator()(const T &x) {
		return reciprocal(x);
	}
};

template <typename T, class OpClass> T transformAndSumCPU(std::vector<T> data, OpClass op)
{
	std::vector<T> temp(data.size());
	std::transform(data.begin(), data.end(), temp.begin(), op);
	return std::accumulate(temp.begin(), temp.end(), (T)0);
}

template <typename T, class OpClass> T transformAndSumGPU(std::vector<T> data, OpClass op)
{

	std::clock_t begin = std::clock();
		thrust::device_vector<T> temp( data.end() - data.begin());
		std::clock_t end = std::clock();

		double elapsed_secs = double(end - begin) / CLOCKS_PER_SEC;
		std::cout<<"Seconds Allocating Slow"<<elapsed_secs<<std::endl;

		begin = std::clock();
			thrust::copy( data.begin(), data.end(),temp.begin());
			end = std::clock();

			elapsed_secs = double(end - begin) / CLOCKS_PER_SEC;
			std::cout<<"Seconds Copying Slow"<<elapsed_secs<<std::endl;

		begin = std::clock();
		thrust::transform(temp.begin(), temp.end(), temp.begin(), op);
		end = std::clock();

		elapsed_secs = double(end - begin) / CLOCKS_PER_SEC;
		std::cout<<"Seonds  transforming Slow"<<elapsed_secs<<std::endl;


		begin = std::clock();
			T result = thrust::reduce(temp.begin(), temp.end());
				end = std::clock();

				elapsed_secs = double(end - begin) / CLOCKS_PER_SEC;
				std::cout<<"Seonds  Reducing Slow"<<elapsed_secs<<std::endl;


			return result;
}


template <typename T, class OpClass> T transformAndSumGPUCustomTempAlloc(std::vector<T> data, OpClass op)
{

	std::clock_t begin = std::clock();
	thrust::device_vector<T, BlazingDeviceAllocator<T> > temp( data.end() - data.begin());
	std::clock_t end = std::clock();

	double elapsed_secs = double(end - begin) / CLOCKS_PER_SEC;
	std::cout<<"Seconds Allocating"<<elapsed_secs<<std::endl;

	begin = std::clock();
		thrust::copy( data.begin(), data.end(),temp.begin());
		end = std::clock();

		elapsed_secs = double(end - begin) / CLOCKS_PER_SEC;
		std::cout<<"Seconds Copying"<<elapsed_secs<<std::endl;

	begin = std::clock();
	thrust::transform(thrust::cuda::par(cachedDeviceAllocator),temp.begin(), temp.end(), temp.begin(), op);
	end = std::clock();

	elapsed_secs = double(end - begin) / CLOCKS_PER_SEC;
	std::cout<<"Seonds  transforming"<<elapsed_secs<<std::endl;
	begin = std::clock();
	T result = thrust::reduce(thrust::cuda::par(cachedDeviceAllocator),temp.begin(), temp.end());
		end = std::clock();

		elapsed_secs = double(end - begin) / CLOCKS_PER_SEC;
		std::cout<<"Seonds  Reducing"<<elapsed_secs<<std::endl;


	return result;
}


template<typename T> void initialize(std::vector<T> &data, unsigned workSize)
{
	/* Initialize the vector */
	for (unsigned i = 0; i < workSize; i++)
		data.push_back( ((T)0.1)*(i+1) );
}
int iterations =  5;
template<typename T> void doCompute(unsigned workSize)
{
	std::vector<T> hostData;

	initialize(hostData, workSize);
	T cpuResults = transformAndSumCPU(hostData, ReciprocalFunctor<T>());

	std::clock_t begin = std::clock();
	T gpuResults;
	for(int i = 0; i < iterations; i++){
		 gpuResults += transformAndSumGPU(hostData, ReciprocalFunctor<T>());
		

	}

	std::clock_t end = std::clock();

	double elapsed_secs = double(end - begin) / CLOCKS_PER_SEC;

	begin = std::clock();

	for(int i = 0; i < iterations; i++){
			gpuResults += transformAndSumGPUCustomTempAlloc(hostData, ReciprocalFunctor<T>());

	}

	end = std::clock();

	double elapsed_secs_custom = double(end - begin) / CLOCKS_PER_SEC;


	std::cout<<"transformAndSumCPU = "<<cpuResults<<std::endl;
	std::cout<<"transformAndSumGPU = "<<gpuResults<<" in thiis many seconds "<<elapsed_secs<<std::endl;
	std::cout<<"transformAndSumGPUCustomTempAlloc = "<<gpuResults<<" in thiis many seconds "<<elapsed_secs_custom<<std::endl;
}



int main(void)
{
	int * x;
	hipMalloc((void **) &x, 100000);
	hipFree(x);

	doCompute<long long> (1024*1024*200);
	return 0;
}
